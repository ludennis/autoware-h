/*
 * Copyright (c) 2014, Nagoya University
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 *
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * * Neither the name of Autoware nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <ndt_matcher/ndt_gpu/registration.h>
#include <ndt_matcher/ndt_gpu/registration_kernel.cuh>

namespace gpu
{

static const int MAX_NEAREST_POINTS_DISTANCE_NUMBER = 60000;

GRegistration::GRegistration()
{
  mMaxIterations = 0;
  mSourcePointsX = mSourcePointsY = mSourcePointsZ = nullptr;
  mSourcePointsNumber = 0;
  mTransformedPointsX = mTransformedPointsY = mTransformedPointsZ = nullptr;
  mNearestPointsDistanceNumber = 0;
  mConverged = false;
  mNumberIteration = 0;
  mTransformationEpsilon = 0;
  mTargetPointCloudUpdated = true;
  mTargetPointsNumber = 0;
  mTargetPointsX = mTargetPointsY = mTargetPointsZ = nullptr;
  mNearestPointsDistanceX =
    AllocateCudaMemory<float>(MAX_NEAREST_POINTS_DISTANCE_NUMBER);
  mNearestPointsDistanceY =
    AllocateCudaMemory<float>(MAX_NEAREST_POINTS_DISTANCE_NUMBER);
  mNearestPointsDistanceZ =
    AllocateCudaMemory<float>(MAX_NEAREST_POINTS_DISTANCE_NUMBER);
  mNearestPointsDistanceDevice =
    AllocateCudaMemory<pcl::PointXYZ>(MAX_NEAREST_POINTS_DISTANCE_NUMBER);
}

void GRegistration::SetTransformationEpsilon(double transformationEpsilon)
{
  mTransformationEpsilon = transformationEpsilon;
}

double GRegistration::GetTransformationEpsilon() const
{
  return mTransformationEpsilon;
}

void GRegistration::SetMaximumIterations(int maximumIterations)
{
  mMaxIterations = maximumIterations;
}

int GRegistration::GetMaximumIterations() const
{
  return mMaxIterations;
}

Eigen::Matrix<float, 4, 4> GRegistration::GetFinalTransformation() const
{
  return mFinalTransformationMatrix;
}

int GRegistration::GetFinalNumIteration() const
{
  return mNumberIteration;
}

bool GRegistration::HasConverged() const
{
  return mConverged;
}

void GRegistration::SetInputSource(pcl::PointCloud<pcl::PointXYZI>::Ptr input)
{
  if (input->size() > 0) {
    mSourcePointsNumber = input->size();

    std::shared_ptr<pcl::PointXYZI> tmp =
      AllocateCudaMemory<pcl::PointXYZI>(mSourcePointsNumber);
    pcl::PointXYZI *pointsFromHost = input->points.data();

    #ifndef __aarch64__
      checkCudaErrors(hipHostRegister(pointsFromHost,
        sizeof(pcl::PointXYZI) * mSourcePointsNumber, hipHostRegisterDefault));
    #endif
    checkCudaErrors(hipMemcpy(tmp.get(),
      pointsFromHost, sizeof(pcl::PointXYZI) * mSourcePointsNumber,
      hipMemcpyHostToDevice));

    mSourcePointsX = AllocateCudaMemory<float>(mSourcePointsNumber);
    mSourcePointsY = AllocateCudaMemory<float>(mSourcePointsNumber);
    mSourcePointsZ = AllocateCudaMemory<float>(mSourcePointsNumber);

    int blockX = (mSourcePointsNumber > BLOCK_SIZE_X) ?
      BLOCK_SIZE_X : mSourcePointsNumber;
    int gridX = (mSourcePointsNumber - 1) / blockX + 1;

    ConvertInputPoints<pcl::PointXYZI><<<gridX, blockX>>>(tmp.get(),
      mSourcePointsX.get(), mSourcePointsY.get(), mSourcePointsZ.get(),
      mSourcePointsNumber);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    mTransformedPointsX = AllocateCudaMemory<float>(mSourcePointsNumber);
    mTransformedPointsY = AllocateCudaMemory<float>(mSourcePointsNumber);
    mTransformedPointsZ = AllocateCudaMemory<float>(mSourcePointsNumber);

    checkCudaErrors(hipMemcpy(mTransformedPointsX.get(),
      mSourcePointsX.get(), sizeof(float) * mSourcePointsNumber,
      hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(mTransformedPointsY.get(),
      mSourcePointsY.get(), sizeof(float) * mSourcePointsNumber,
      hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(mTransformedPointsZ.get(),
      mSourcePointsZ.get(), sizeof(float) * mSourcePointsNumber,
      hipMemcpyDeviceToDevice));

    #ifndef __aarch64__
      checkCudaErrors(hipHostUnregister(pointsFromHost));
    #endif
  }
}

void GRegistration::SetInputSource(pcl::PointCloud<pcl::PointXYZ>::Ptr input)
{
  if (input->size() > 0) {
    mSourcePointsNumber = input->size();

    std::shared_ptr<pcl::PointXYZ> tmp =
      AllocateCudaMemory<pcl::PointXYZ>(mSourcePointsNumber);
    pcl::PointXYZ *pointsFromHost = input->points.data();

    #ifndef __aarch64__
      checkCudaErrors(hipHostRegister(pointsFromHost,
        sizeof(pcl::PointXYZ) * mSourcePointsNumber, hipHostRegisterDefault));
    #endif
    checkCudaErrors(hipMemcpy(tmp.get(),
      pointsFromHost, sizeof(pcl::PointXYZ) * mSourcePointsNumber,
      hipMemcpyHostToDevice));

    mSourcePointsX = AllocateCudaMemory<float>(mSourcePointsNumber);
    mSourcePointsY = AllocateCudaMemory<float>(mSourcePointsNumber);
    mSourcePointsZ = AllocateCudaMemory<float>(mSourcePointsNumber);

    int blockX = (mSourcePointsNumber > BLOCK_SIZE_X) ?
      BLOCK_SIZE_X : mSourcePointsNumber;
    int gridX = (mSourcePointsNumber - 1) / blockX + 1;

    ConvertInputPoints<pcl::PointXYZ><<<gridX, blockX>>>(tmp.get(),
      mSourcePointsX.get(), mSourcePointsY.get(),
      mSourcePointsZ.get(), mSourcePointsNumber);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    mTransformedPointsX = AllocateCudaMemory<float>(mSourcePointsNumber);
    mTransformedPointsY = AllocateCudaMemory<float>(mSourcePointsNumber);
    mTransformedPointsZ = AllocateCudaMemory<float>(mSourcePointsNumber);

    checkCudaErrors(hipMemcpy(mTransformedPointsX.get(),
      mSourcePointsX.get(), sizeof(float) * mSourcePointsNumber,
      hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(mTransformedPointsY.get(),
      mSourcePointsY.get(), sizeof(float) * mSourcePointsNumber,
      hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(mTransformedPointsZ.get(),
      mSourcePointsZ.get(), sizeof(float) * mSourcePointsNumber,
      hipMemcpyDeviceToDevice));

    #ifndef __aarch64__
      checkCudaErrors(hipHostUnregister(pointsFromHost));
    #endif
  }
}

void GRegistration::SetInputTarget(pcl::PointCloud<pcl::PointXYZI>::Ptr input)
{
  if (input->size() > 0) {
    mTargetPointsNumber = input->size();

    std::shared_ptr<pcl::PointXYZI> tmp =
      AllocateCudaMemory<pcl::PointXYZI>(mTargetPointsNumber);
    pcl::PointXYZI *pointsFromHost = input->points.data();

    #ifndef __aarch64__
      checkCudaErrors(hipHostRegister(pointsFromHost,
        sizeof(pcl::PointXYZI) * mTargetPointsNumber, hipHostRegisterDefault));
    #endif
    checkCudaErrors(hipMemcpy(tmp.get(),
      pointsFromHost, sizeof(pcl::PointXYZI) * mTargetPointsNumber,
      hipMemcpyHostToDevice));

    mTargetPointsX = AllocateCudaMemory<float>(mTargetPointsNumber);
    mTargetPointsY = AllocateCudaMemory<float>(mTargetPointsNumber);
    mTargetPointsZ = AllocateCudaMemory<float>(mTargetPointsNumber);

    int blockX = (mTargetPointsNumber > BLOCK_SIZE_X) ?
      BLOCK_SIZE_X : mTargetPointsNumber;
    int gridX = (mTargetPointsNumber - 1) / blockX + 1;

    ConvertInputPoints<pcl::PointXYZI><<<gridX, blockX>>>(tmp.get(),
      mTargetPointsX.get(), mTargetPointsY.get(), mTargetPointsZ.get(),
      mTargetPointsNumber);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    #ifndef __aarch64__
      checkCudaErrors(hipHostUnregister(pointsFromHost));
    #endif
  }
}

void GRegistration::SetInputTarget(pcl::PointCloud<pcl::PointXYZ>::Ptr input)
{
  PROFILE_CUDA_MEMORY_BEGIN(GRegistration_setInputTarget);
  if (input->size() > 0) {
    mTargetPointsNumber = input->size();

    std::shared_ptr<pcl::PointXYZ> tmp =
      AllocateCudaMemory<pcl::PointXYZ>(mTargetPointsNumber);
    pcl::PointXYZ *pointsFromHost = input->points.data();

    #ifndef __aarch64__
      checkCudaErrors(hipHostRegister(pointsFromHost,
        sizeof(pcl::PointXYZ) * mTargetPointsNumber, hipHostRegisterDefault));
    #endif
    checkCudaErrors(hipMemcpy(tmp.get(),
      pointsFromHost, sizeof(pcl::PointXYZ) * mTargetPointsNumber,
      hipMemcpyHostToDevice));

    mTargetPointsX = AllocateCudaMemory<float>(mTargetPointsNumber);
    mTargetPointsY = AllocateCudaMemory<float>(mTargetPointsNumber);
    mTargetPointsZ = AllocateCudaMemory<float>(mTargetPointsNumber);

    int blockX = (mTargetPointsNumber > BLOCK_SIZE_X) ?
      BLOCK_SIZE_X : mTargetPointsNumber;
    int gridX = (mTargetPointsNumber - 1) / blockX + 1;

    ConvertInputPoints<pcl::PointXYZ><<<gridX, blockX>>>(tmp.get(),
      mTargetPointsX.get(), mTargetPointsY.get(), mTargetPointsZ.get(),
      mTargetPointsNumber);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    #ifndef __aarch64__
      checkCudaErrors(hipHostUnregister(pointsFromHost));
    #endif
  }
  PROFILE_CUDA_MEMORY_END(GRegistration_setInputTarget);
}

void GRegistration::SetInputNearestPointsDistanceSource(
  pcl::PointCloud<pcl::PointXYZ>::Ptr input)
{
  if (input->size() > 0) {
    mNearestPointsDistanceNumber = input->size();
    pcl::PointXYZ *pointsFromHost = input->points.data();

    #ifndef __aarch64__
      checkCudaErrors(hipHostRegister(pointsFromHost,
        sizeof(pcl::PointXYZ) * mNearestPointsDistanceNumber,
        hipHostRegisterDefault));
    #endif

    if (mNearestPointsDistanceNumber > MAX_NEAREST_POINTS_DISTANCE_NUMBER)
    {
      throw std::runtime_error("Number of points, used for semantic "
      "segmentation, exceeds the limit.");
    }

    checkCudaErrors(hipMemcpy(mNearestPointsDistanceDevice.get(),
      pointsFromHost, sizeof(pcl::PointXYZ) * mNearestPointsDistanceNumber,
      hipMemcpyHostToDevice));

    int blockX = (mNearestPointsDistanceNumber > BLOCK_SIZE_X) ?
      BLOCK_SIZE_X : mNearestPointsDistanceNumber;
    int gridX = (mNearestPointsDistanceNumber - 1) / blockX + 1;

    ConvertInputPoints<pcl::PointXYZ><<<gridX, blockX>>>(
      mNearestPointsDistanceDevice.get(), mNearestPointsDistanceX.get(),
      mNearestPointsDistanceY.get(), mNearestPointsDistanceZ.get(),
      mNearestPointsDistanceNumber);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    #ifndef __aarch64__
      checkCudaErrors(hipHostUnregister(pointsFromHost));
    #endif
  }
}

void GRegistration::Align(const Eigen::Matrix<float, 4, 4> &guess)
{
  mConverged = false;
  mFinalTransformationMatrix =
    mTransformationMatrix = mPreviousTransformationMatrix =
    Eigen::Matrix<float, 4, 4>::Identity();

  ComputeTransformation(guess);
}

void GRegistration::ComputeTransformation(const Eigen::Matrix<float, 4, 4> &guess)
{
  printf("Unsupported by Registration\n");
}

} // namespace gpu
